#include "hip/hip_runtime.h"
#include "ParseGraph.hxx"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#define MAX 1024

using namespace std;

__global__
void isInfected(long long *person, long long *solution, bool *visited, bool *status, long long n) {
    long long ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= n) return;
    visited[person[ind] - 1] = true;
    if (status[person[ind] - 1]) {
        *solution = person[ind] - 1;
    }
}


int main(void) {
    readNames();
    makeGraph();
    bool *h_Status = new bool[Status.size()];
    copy(Status.begin(), Status.end(), h_Status);
    bool *d_Status;
    hipMalloc((void**)&d_Status, sizeof(bool) * Status.size());
    hipMemcpy(d_Status, h_Status, sizeof(bool) * Status.size(), hipMemcpyHostToDevice);
    delete h_Status;

    vector<long long> neighbors;
    bool *visited;
    long long *solution;
    hipMallocManaged((void**)&solution, sizeof(long long));
    hipMallocManaged((void**)&visited, sizeof(bool) * People.size());
    *solution = -1;
    
    cout << "Enter ID of person: ";
    long long id; cin >> id;

    if (id < 1 || id >= People.size()) {
        cerr << "\nID out of range\n";
        return -1;
    }
    neighbors.push_back(id);

    Timer timer;
    while (!neighbors.empty() && *solution == -1) {
        long long *d_neighbors;
        hipMalloc((void**)&d_neighbors, sizeof(long long) * neighbors.size());
        hipMemcpy(d_neighbors, &neighbors[0], sizeof(long long) * neighbors.size(), hipMemcpyHostToDevice);
        isInfected<<<neighbors.size() / MAX + 1, MAX>>>(d_neighbors, solution, visited, d_Status, neighbors.size());
        hipDeviceSynchronize();
        hipFree(d_neighbors);
        
        if (*solution != -1) {
            cout << "Closest Infected Person: \n" << People[*solution].info() << endl;
            break;
        }
        
        vector<long long> newNeighbors;
        for (auto neighbor: neighbors) {
            copy_if(Graph[neighbor - 1].begin(), Graph[neighbor - 1].end(), back_inserter(newNeighbors), [visited](long long id) {
                return !visited[id];
            });
        }
        neighbors.clear();
        neighbors = newNeighbors;
    }
    double time = timer.Stop();

    if (*solution == -1) {
        cout << "\nPerson has not come in contact with an infected person\n";
    }

    cout << "\nTime Take: " << time << endl;

    hipFree(solution);
    hipFree(visited);
    
    return 0;
}