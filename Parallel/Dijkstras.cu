#include "hip/hip_runtime.h"
#include "ParseGraph.hxx"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#define MAX 1024

using namespace std;

__global__
void isInfected(long long *person, long long *solution, bool *visited, bool *status, long long n) {
    long long ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= n) return;
    visited[person[ind] - 1] = true;
    if (status[person[ind] - 1]) {
        *solution = person[ind] - 1;
    }
}


int main(void) {
    readNames();
    makeGraph();
    bool *h_Status = new bool[Status.size()];
    copy(Status.begin(), Status.end(), h_Status);
    bool *d_Status;
    hipMalloc((void**)&d_Status, sizeof(bool) * Status.size());
    hipMemcpy(d_Status, h_Status, sizeof(bool) * Status.size(), hipMemcpyHostToDevice);
    delete h_Status;

    while (true) {
        vector<long long> neighbors;
        bool *visited;
        long long *solution;
        hipMallocManaged((void**)&solution, sizeof(long long));
        hipMallocManaged((void**)&visited, sizeof(bool) * People.size());
        *solution = -1;
        
        cout << "Enter ID of person: ";
        long long id; cin >> id;

        if (id < 1 || id >= People.size()) {
            cerr << "\nID out of range\n";
            return -1;
        }
        neighbors.push_back(id);

        cout << "\nProcessing...";
        Timer timer, op;
        while (!neighbors.empty() && *solution == -1) {
            if (op.GetTime() >= 0.1) {
                cout << "\rEntries Processed: " << count_if(visited, visited + People.size(), [](bool status) {
                    return status;
                });
                cout.flush();
                op.Reset();
            }

            long long *d_neighbors;
            hipMalloc((void**)&d_neighbors, sizeof(long long) * neighbors.size());
            hipMemcpy(d_neighbors, &neighbors[0], sizeof(long long) * neighbors.size(), hipMemcpyHostToDevice);
            isInfected<<<neighbors.size() / MAX + 1, MAX>>>(d_neighbors, solution, visited, d_Status, neighbors.size());
            hipDeviceSynchronize();
            hipFree(d_neighbors);
            
            if (*solution != -1) {
                cout << "\rEntries Processed: " << count_if(visited, visited + People.size(), [](bool status) {
                    return status;
                }) 
                    << "... Done" << endl;
                cout << "Closest Infected Person: \n" << People[*solution].info() << endl;
                break;
            }
            
            vector<long long> newNeighbors;
            for (auto neighbor: neighbors) {
                copy_if(Graph[neighbor - 1].begin(), Graph[neighbor - 1].end(), back_inserter(newNeighbors), [visited](long long id) {
                    return !visited[id];
                });
            }
            neighbors.clear();
            neighbors = newNeighbors;
        }
        double time = timer.Stop();

        if (*solution == -1) {
            cout << "\rEntries Processed: " << count_if(visited, visited + People.size(), [](bool status) {
                return status;
            }) 
                << "... Done" << endl;
            cout << "\nPerson has not come in contact with an infected person\n";
        }

        cout << "\nTime Taken: " << time << endl;

        hipFree(solution);
        hipFree(visited);    
    }
    return 0;
}